
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#define MAX_ROTATIONS  80
#define GENERAL_MEMORY_PROBLEM printf( "You do not have enough memory ([m|re]alloc failure)\nDying\n\n" ) ; exit( EXIT_FAILURE ) ;

struct Atom{
    int charge;
    int name;
};
struct Residue
{
    int n;
    struct Atom *atoms;

};

__global__ void foo2(struct Residue *r)
{
    
    int i=threadIdx.y;
    int j= threadIdx.x;
    r[i].atoms[j].name=j;
    r[j].atoms[j].charge=j;
    printf("reached\n");
    printf("%d %d\n",r[i].atoms[j].name,r[j].atoms[j].charge);

}



int main()
{
    struct Residue r[2],*rc;

        hipMalloc((void**)&(r[0].atoms),2*sizeof(struct Atom));
        hipDeviceSynchronize();
        hipMalloc((void**)&(r[1].atoms),2*sizeof(struct Atom));
        hipDeviceSynchronize();
        hipMalloc((void**)&rc,2*sizeof(struct Residue));
        hipDeviceSynchronize();
        hipMemcpy(rc,r,2*sizeof(struct Residue),hipMemcpyHostToDevice);
        hipDeviceSynchronize();

    
     printf("hello whatsup");

    
    

    dim3 threadsperblock(2,2);
    foo2<<<1,threadsperblock>>>(rc);
    hipDeviceSynchronize();
  


    

}

