
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#define MAX_ROTATIONS  80
#define GENERAL_MEMORY_PROBLEM printf( "You do not have enough memory ([m|re]alloc failure)\nDying\n\n" ) ; exit( EXIT_FAILURE ) ;

struct electrostatics
{
    int charge;

};
__global__ void testfunc(struct electrostatics *a)
{
    int i = threadIdx.x;
    a[i].charge=2+i;
}

int main()
{
    struct electrostatics *a,b[2];
    hipMalloc(&a,2*sizeof(electrostatics));
    testfunc<<<1,2>>>(a);
    hipDeviceSynchronize();
    hipMemcpy(b,a,2*sizeof(electrostatics),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("%d %d",b[0].charge,b[1].charge);

}

