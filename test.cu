#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define N 256
__global__ void add(int *a, int *b, int *c)
{
 int tID = threadIdx.x;
 if (tID < N)
 {
 c[tID] = a[tID] + b[tID];
 }
}
int main()
{
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, N*sizeof(int));
 hipMalloc((void **) &dev_b, N*sizeof(int));
 hipMalloc((void **) &dev_c, N*sizeof(int));
 // Fill Arrays
 for (int i = 0; i < N; i++)
 {
 a[i] = i,
 b[i] = 1;
 }
 clock_t s,e;
 s=clock();
 double time;

 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
 add<<<1,N>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
 for (int i = 0; i < N; i++)
 {
 printf("%d + %d = %d\n", a[i], b[i], c[i]);
 }
 e=clock();
 time=e-s;
 printf("%f is the time taken",time);

 return 0;
}
