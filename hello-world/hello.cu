
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
__global__ void cuda_hello(){
    int x=threadIdx.x;
    int y=blockIdx.x;
    printf("Hello World from GPU! %d %d \n",x,y);
 
}

int main() {
    clock_t s,e;
    double time;
    s=clock();


    for (int i = 0; i < 9600; i++)
    {
        printf("Hello World from CPU!\n");

    }
    e=clock();
    time=e-s;
    double ti=time;
    printf("\n%f\n\n",time);
    s=clock();
    cuda_hello<<<100,96>>>();
    hipDeviceSynchronize();
    e=clock();
    time=e-s;
    printf("\n%f  %f\n",time,ti);
    return 0;
}