
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

__global__ void Adderf(hipfftReal *a,float b,float c)
{
    *a=b+c;
    
}
int main()
{
    hipfftReal *a,b;
    float a1=2.0,b1=3.5;
    if(hipMalloc((void**)&a,sizeof(hipfftReal))==hipErrorOutOfMemory)printf("wgharts");
  
    Adderf<<<1,1>>>(a,a1,b1);
    hipDeviceSynchronize();
    hipMemcpy(&b,a,sizeof(hipfftReal),hipMemcpyDeviceToHost);

    printf("%f",b);

}