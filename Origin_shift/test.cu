#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
__global__ void foo(int *s)
{
    int idx = threadIdx.x;
    int sum = 0;
    for (int i = idx; i < ; i += blockSize)
        sum += a[i];
    __shared__ int r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (idx<size)
            r[idx] += r[idx+size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

int main()
{
    int *sum,a=0;
    hipMalloc((void**)&sum,sizeof(int));
    hipMemcpy(sum,&a,sizeof(int),hipMemcpyHostToDevice);
    foo<<<1,10>>>(sum);
    hipDeviceSynchronize();
    hipMemcpy(&a,sum,sizeof(int),hipMemcpyDeviceToHost);
    printf("%d",a);


}