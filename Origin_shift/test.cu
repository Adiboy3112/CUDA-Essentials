
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

    hipfftReal *a,*b;
    hipfftComplex *c;


__global__ void Adderf(hipfftReal *a,float b,float c)
{
    *a=b+c;
    
}
void foo(hipfftReal *a)
{
    float a1=2.0,b1=3.5;
    Adderf<<<1,1>>>(a,a1,b1);
}
int main()
{
    if(hipMalloc((void**)&a,sizeof(hipfftReal))==hipErrorOutOfMemory)printf("wgharts");
    b=(hipfftReal *)malloc(sizeof(hipfftReal));
  
    foo(a);
    hipDeviceSynchronize();
    hipMemcpy(b,a,sizeof(hipfftReal),hipMemcpyDeviceToHost);
    c = (hipfftComplex *)b;
    c->y=1.5;

    printf("%f",(c->x+c->y));

}