
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#define MAX_ROTATIONS  80
#define GENERAL_MEMORY_PROBLEM printf( "You do not have enough memory ([m|re]alloc failure)\nDying\n\n" ) ; exit( EXIT_FAILURE ) ;
typedef struct __align__{
	int	*z_twist ;
	int	*theta ;
	int	*phi ;
} Angle;
__global__ void testfunc(Angle Angles)
{
    int i=threadIdx.x;
    Angles.z_twist[i] = i ;
    Angles.theta[i]   = 0 ;
    Angles.phi[i]     = 1 ;
    printf("%d\n",Angles.z_twist[i]);



}
int main()
{
    Angle Angles,AnglesonGPU;
  Angles.z_twist = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int ));
  Angles.theta   = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int ));
  Angles.phi     = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int )); 
  printf("%zu\n\n",sizeof(Angles));

  hipMalloc((void**)&AnglesonGPU.z_twist,MAX_ROTATIONS * sizeof( int ));
  hipMalloc((void**)&AnglesonGPU.theta,MAX_ROTATIONS * sizeof( int ));
  hipMalloc((void**)&AnglesonGPU.phi,MAX_ROTATIONS * sizeof( int ));
  testfunc<<<1,MAX_ROTATIONS>>>(AnglesonGPU);
  hipDeviceSynchronize();
  int x=MAX_ROTATIONS*sizeof(int);
  hipMemcpy(Angles.z_twist, AnglesonGPU.z_twist,x,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < MAX_ROTATIONS; i++)
  {
    printf("%d\n",Angles.z_twist[i]);
  }
  





}
