
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define MAX_ROTATIONS  1000
#define GENERAL_MEMORY_PROBLEM printf( "You do not have enough memory ([m|re]alloc failure)\nDying\n\n" ) ; exit( EXIT_FAILURE ) ;
typedef struct __align__{
	int	n ;
	int	*z_twist ;
	int	*theta ;
	int	*phi ;
} Angle;
// __global__ void testfunc()
// {

// }
int main()
{
    Angle Angles,AnglesonGPU;
    if( (Angles.z_twist = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int ) ) ) &&
      ( Angles.theta   = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int ) ) ) &&
      ( Angles.phi     = ( int * ) malloc ( MAX_ROTATIONS * sizeof( int ) ) ) ) {
  } else {
    GENERAL_MEMORY_PROBLEM
  }
  hipMalloc((void**)&AnglesonGPU,sizeof(Angle));
  hipMalloc((void**)&AnglesonGPU.z_twist,MAX_ROTATIONS * sizeof( int ));
  hipMalloc((void**)&AnglesonGPU.theta,MAX_ROTATIONS * sizeof( int ));
  hipMalloc((void**)&AnglesonGPU.phi,MAX_ROTATIONS * sizeof( int ));
  printf("%zu",sizeof(Angles.z_twist));


}
